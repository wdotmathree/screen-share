
#include <hip/hip_runtime.h>
#include <iostream>

unsigned int *dev_in, *dev_out;

__global__ void downscale(unsigned int *in, unsigned int *out, int sw, int sh, int dw, int dh) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x2 = x * sw / dw;
	int y2 = y * sh / dh;
	int x3 = (x + 1) * sw / dw;
	int y3 = (y + 1) * sh / dh;
	int r = 0, g = 0, b = 0;
	for (int i = x2; i < x3; i++) {
		for (int j = y2; j < y3; j++) {
			unsigned int pix = in[i + j * sw];
			r += pix & 0xff;
			g += (pix >> 8) & 0xff;
			b += (pix >> 16) & 0xff;
		}
	}
	r /= (x3 - x2) * (y3 - y2);
	g /= (x3 - x2) * (y3 - y2);
	b /= (x3 - x2) * (y3 - y2);
	out[x + y * dw] = r | (g << 8) | (b << 16);
}

extern "C" int scale_img(unsigned int *in, unsigned int *out, int sw, int sh, int dw, int dh) {
	if (dev_in == NULL) {
		hipMalloc((void **)&dev_in, sw * sh * 4);
		hipMalloc((void **)&dev_out, dw * dh * 4);
	}
	hipMemcpy(dev_in, in, sw * sh * 4, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks(dw / threads.x, dh / threads.y);
	downscale<<<blocks, threads>>>(dev_in, dev_out, sw, sh, dw, dh);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << hipGetErrorString(err) << std::endl;
		return -1;
	}
	hipMemcpy(out, dev_out, dw * dh * 4, hipMemcpyDeviceToHost);
	return 0;
}
